// standard c headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <cblas.h>
#include <float.h>

// own c headers
#include "common.h"
#include "global.h"
#include "pw2d_comp.h"
#include "common_utils.h"


int main(int argc, char **argv)
{

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("\nTiming 2D pointwise at ");
  printf("device %d: %s \n\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  printf("Performs the following Timings:\n \n - Add along axis on HOST and DEVICE\n\n_________________________________________________\n");


  srand(seconds());   // Initialization, should only be called once.
  double start;

  // time pointwise_combine problem
  double t,t1,t2,t3,t4;

  double *dev_mat_in,*dev_mat_out,*dev_vec;
  double *mat_in,*mat_out,*vec;



  FILE *fp_ew2d = fopen("analysis/fp_ew2d.txt", "w");
  fprintf(fp_ew2d,"N_ROWS\tN_COLS\tMRTIME_onDEV\tMRTIME_HOST\tCT_DtH\tCT_HtD\n");



  for(int cols=1;cols<=(1<<13);cols<<=1){
    for(int rows=1;rows<=(1<<13);rows<<=1){

      t1=t2=t3=t4=DBL_MAX;

      int size_mat_in=cols*rows;
      int size_mat_out=size_mat_in;
      mat_in=(double *)malloc(size_mat_in*sizeof(double));
      mat_out=(double *)malloc(size_mat_out*sizeof(double));
      vec=(double *)malloc(cols*sizeof(double));

      for(int i =0;i<size_mat_in;i++) mat_in[i]=((double)rand()/(double)RAND_MAX);
      for(int i =0;i<cols;i++) vec[i]=((double)rand()/(double)RAND_MAX);

      CHECK(hipMalloc((void**)&dev_mat_in, size_mat_in*sizeof(double)));
      CHECK(hipMalloc((void**)&dev_mat_out, size_mat_out*sizeof(double)));
      CHECK(hipMalloc((void**)&dev_vec, cols*sizeof(double)));

      copy_host_to_device_double(mat_in,dev_mat_in,size_mat_in);
      copy_host_to_device_double(vec,dev_vec,cols);

      for (int i=0;i<5;i++){
        start=seconds();
        func_along_axis_onDev<double>(add_functor<double>(),dev_mat_in,dev_vec,dev_mat_out, rows,cols, 0, cols);
        t=seconds()-start;
        t1=t<t1?t:t1;

        start=seconds();
        func_along_axis_cpu<double>(add_functor<double>(),mat_in,vec,mat_out, rows,cols, 0, cols);
        t=seconds()-start;
        t2=t<t2?t:t2;

        start=seconds();
        copy_device_to_host_double(dev_mat_in,mat_in,size_mat_in);
        copy_device_to_host_double(dev_vec,vec,cols);
        t=seconds()-start;
        t3=t<t3?t:t3;

        start=seconds();
        copy_host_to_device_double(mat_out,dev_mat_out,size_mat_out);
        t=seconds()-start;
        t4=t<t4?t:t4;

      }
      fprintf(fp_ew2d,"%d\t%d\t%e\t%e\t%e\t%e\n",rows,cols,t1,t2,t3,t4);

      CHECK(hipFree(dev_mat_in));
      CHECK(hipFree(dev_mat_out));
      CHECK(hipFree(dev_vec));
      free(mat_in);
      free(mat_out);
      free(vec);
    }
  }
  fclose (fp_ew2d);

}
