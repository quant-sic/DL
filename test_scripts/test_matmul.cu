#include "hip/hip_runtime.h"
// standard c headers
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <float.h>

// own c headers
#include "common.h"
#include "global.h"
#include "test_matrix_operator.h"
#include "mat_mul.h"
#include "common_utils.h"
#include "pw_comp.h"


// define thresholds
#define MATMUL_COMP(K) (sqrt(2*K)*DBL_EPSILON)




int main(int argc, char **argv)
{

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Testing MatrixMultiplication at ");
  printf("device %d: %s \n\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));

  printf("Performs the following checks:\n\n - matMul on HOST and matMul_gpu1, matMul_gpu2 matMul_gpu_dsm, matMul_gpu_dsm_coa, matMul_cublas on Device\n - HOST and DEVICE same result and All yield same result\n");
  printf(" - ADD, Scale and Matrix Multiply compatible (Distributivgesetz)\n");
  printf(" - ONE, Scale and Matrix Multiply + Transpose and Transpose compatible\n");
  printf(" - Multiply + Transpose and Transpose compatible\n");
  printf("\n_________________________________________________\n");


  srand(seconds());   // Initialization, should only be called once.
  double *A,*B,*B_T,*B_T_T,*C1,*C2,*C3,*C4,*C5,*C6,*C7,*C8,*A_T;
  double *A2,*B2,*A3,*B3,*As,*Bs,*A2s,*B2s,*one_m;
  int same_result;

  for(int M=1;M<=256;M*=2){
    for(int N=1;N<=256;N*=2){
      for(int K=1;K<=256;K*=2){

        int A_nelem=M*K;
        int B_nelem=K*N;
        int C_nelem=M*N;

        A = (double *)malloc(A_nelem*sizeof(double));
        B = (double *)malloc(B_nelem*sizeof(double));
        A2 = (double *)malloc(A_nelem*sizeof(double));
        B2 = (double *)malloc(B_nelem*sizeof(double));
        As = (double *)malloc(A_nelem*sizeof(double));
        Bs = (double *)malloc(B_nelem*sizeof(double));
        A2s = (double *)malloc(A_nelem*sizeof(double));
        B2s = (double *)malloc(B_nelem*sizeof(double));
        A3 = (double *)malloc(A_nelem*sizeof(double));
        B3 = (double *)malloc(B_nelem*sizeof(double));
        A_T = (double *)malloc(A_nelem*sizeof(double));
        B_T = (double *)malloc(B_nelem*sizeof(double));
        B_T_T = (double *)malloc(B_nelem*sizeof(double));
        C1 = (double *)malloc(C_nelem*sizeof(double));
        C2 = (double *)malloc(C_nelem*sizeof(double));
        C3 = (double *)malloc(C_nelem*sizeof(double));
        C4 = (double *)malloc(C_nelem*sizeof(double));
        C5 = (double *)malloc(C_nelem*sizeof(double));
        C6 = (double *)malloc(C_nelem*sizeof(double));
        C7 = (double *)malloc(C_nelem*sizeof(double));
        C8 = (double *)malloc(C_nelem*sizeof(double));


        create_random_matrix(A,A_nelem,0,5);
        create_random_matrix(B,B_nelem,0,5);



        mat_mul_cpu<double>(A, B,M,N,K,C1);
        mat_mul_coa_onDev<double>(A, B,M,N,K,C3);
        mat_mul_tr_onDev<double>(A, B, NORMAL,NORMAL, M, K, K, N, C7);



        same_result=1;
        same_result*=double_equal(C1,C3,C_nelem,MATMUL_COMP(K));
        same_result*=double_equal(C1,C7,C_nelem,MATMUL_COMP(K));



        if (!same_result){
            printf("For M:%d,N:%d,K:%d Methods do not yield the same result\n",M,N,K);
            print_out_matrix(C1,M,N);
            print_out_matrix(C3,M,N);
            print_out_matrix(C7,M,N);


            return EXIT_FAILURE;
        }


        // ____________________________________________________________________________
        // check if double transposing yields original array
        mat_transpose_onDev<double>(B, B_T, K, N);
        mat_transpose_onDev<double>(B_T, B_T_T, N, K);
        same_result=double_equal(B,B_T_T,B_nelem,DBL_EPSILON);

        if (!same_result){
           printf("For M:%d,N:%d,K:%d Double Transposing does not yield original Matrix\n",M,N,K);
           print_out_matrix(B,K,N);
           print_out_matrix(B_T_T,K,N);
           return EXIT_FAILURE;
        }

        // _____________________________________________________________________________
        // check add matrixmultiply consistency
        int max=10;
        create_random_matrix(A2,A_nelem,0,5);
        create_random_matrix(B2,B_nelem,0,5);
        double alpha,beta,gamma,delta;

        // get scalars
        alpha=(max*(double)rand()/(double)RAND_MAX);
        beta=(max*(double)rand()/(double)RAND_MAX);
        gamma=(max*(double)rand()/(double)RAND_MAX);
        delta=(max*(double)rand()/(double)RAND_MAX);

        // first scale then add then multiply variant
        apply_pointwise_cpu<double>(A,As,A_nelem,scale_functor<double>(alpha));
        apply_pointwise_cpu<double>(A2,A2s,A_nelem,scale_functor<double>(beta));
        apply_pointwise_cpu<double>(B,Bs,B_nelem,scale_functor<double>(gamma));
        apply_pointwise_cpu<double>(B2,B2s,B_nelem,scale_functor<double>(delta));

        combine_pointwise_cpu<double>(As,A2s,A3,A_nelem,add_functor<double>());
        combine_pointwise_cpu<double>(Bs,B2s,B3,B_nelem,add_functor<double>());

        mat_mul_tr_onDev<double>(A3, B3, NORMAL,NORMAL, M, K, K, N, C1);

        // first multiply then scale then add variant
        mat_mul_tr_onDev<double>(A, B, NORMAL,NORMAL, M, K, K, N, C2);
        mat_mul_tr_onDev<double>(A2, B2, NORMAL,NORMAL, M, K, K, N, C3);
        mat_mul_tr_onDev<double>(A, B2, NORMAL,NORMAL, M, K, K, N, C4);
        mat_mul_tr_onDev<double>(A2, B, NORMAL,NORMAL, M, K, K, N, C5);

        apply_pointwise_cpu<double>(C2,C2,C_nelem,scale_functor<double>(alpha*gamma));
        apply_pointwise_cpu<double>(C3,C3,C_nelem,scale_functor<double>(beta*delta));
        apply_pointwise_cpu<double>(C4,C4,C_nelem,scale_functor<double>(alpha*delta));
        apply_pointwise_cpu<double>(C5,C5,C_nelem,scale_functor<double>(beta*gamma));

        combine_pointwise_cpu<double>(C3,C2,C6,C_nelem,add_functor<double>());
        combine_pointwise_cpu<double>(C4,C6,C6,C_nelem,add_functor<double>());
        combine_pointwise_cpu<double>(C5,C6,C6,C_nelem,add_functor<double>());

        // check for equal result
        if (!double_equal(C6,C1,C_nelem,sqrt(4*(1+2*K))*DBL_EPSILON)){
           printf("For M:%d,N:%d,K:%d ADD and MM not compatible\n",M,N,K);
           printf("%e ; %e\n",sqrt(4*(1+2*K))*DBL_EPSILON,max_abs_diff(C1,C6,C_nelem) );
           return EXIT_FAILURE;
        }


        // check if scaled One is consistent with multiplication and scale and transpose matrix consistent with multiply transpose
        if(M==N && N==K){
            one_m = (double *)malloc(N*N*sizeof(double));

            double alpha=(double)(max*(double)rand()/(double)RAND_MAX);

            ONE_Matrix(one_m,N,alpha);

            mat_transpose_onDev<double>(B, B_T, K, N);
            apply_pointwise_cpu<double>(B_T,B_T,B_nelem,scale_functor<double>(alpha));

            mat_mul_tr_onDev<double>(one_m, B, NORMAL,TRANSPOSED, N, N, N, N, C1);

            mat_transpose_onDev<double>(A, A_T, M, K);
            apply_pointwise_cpu<double>(A_T,A_T,A_nelem,scale_functor<double>(alpha));
            mat_mul_tr_onDev<double>(A, one_m, TRANSPOSED,NORMAL, N, N, N, N, C2);

            // check for equal result
            if (!double_equal(B_T,C1,B_nelem,MATMUL_COMP(N))){
               printf("For M:%d,N:%d,K:%d Transpose B and scale and MM_tr not compatible\n",M,N,K);
               printf("%e ; %e\n",10*sqrt(4*K)*DBL_EPSILON,max_abs_diff(C1,C6,C_nelem) );
               return EXIT_FAILURE;
            }
            if (!double_equal(A_T,C2,B_nelem,MATMUL_COMP(N))){
               printf("For M:%d,N:%d,K:%d Transpose A and scale and MM_tr not compatible\n",M,N,K);
               printf("%e ; %e\n",10*sqrt(4*K)*DBL_EPSILON,max_abs_diff(C1,C6,C_nelem) );
               return EXIT_FAILURE;
            }
            free(one_m);
        }


        //checks tranpose in combination with matmul_sm_tr /-ind
        if(M==N){

          C1 = (double *)malloc(M*M*sizeof(double));
          C2 = (double *)malloc(M*M*sizeof(double));
          C7 = (double *)malloc(M*M*sizeof(double));

          C3 = (double *)malloc(K*K*sizeof(double));
          C4 = (double *)malloc(K*K*sizeof(double));
          C8 = (double *)malloc(K*K*sizeof(double));


          mat_transpose_onDev<double>(A, A_T, M, K);

          mat_mul_coa_onDev<double>(A2, A_T,M,M,K,C1);
          mat_mul_tr_onDev<double>(A2, A, NORMAL,TRANSPOSED, M, K, K, M,C2);

          if (!double_equal(C1,C2,M*M,MATMUL_COMP(K))){
             printf("For M:%d,N:%d,K:%d A*A_T same result %d\n",M,N,K);
             return EXIT_FAILURE;
          }

          mat_mul_coa_onDev<double>(A_T, A2,K,K,M,C3);
          mat_mul_tr_onDev<double>(A, A2, TRANSPOSED,NORMAL, K, M, M, K,C4);

          if (!double_equal(C3,C4,K*K,MATMUL_COMP(M))){
             printf("For M:%d,N:%d,K:%d A*A_T not same result\n",M,N,K);
             return EXIT_FAILURE;
          }

          C5 = (double *)malloc(K*K*sizeof(double));
          C6 = (double *)malloc(K*K*sizeof(double));

          mat_transpose_onDev<double>(B, B_T, K, N);
          mat_mul_coa_onDev<double>(A_T, B_T,K,K,M,C5);
          mat_mul_tr_onDev<double>(A, B, TRANSPOSED,TRANSPOSED, K, M, M, K,C6);

          if (!double_equal(C5,C6,K*K,MATMUL_COMP(M))){
             printf("For M:%d,N:%d,K:%d A_T*B_T not same result\n",M,N,K);
             return EXIT_FAILURE;
          }
        }



        free(A);
        free(B);
        free(A2);
        free(B2);
        free(As);
        free(Bs);
        free(A2s);
        free(B2s);
        free(A3);
        free(B3);
        free(C1);
        free(C2);
        free(C3);
        free(C4);
        free(C5);
        free(C6);
        free(C7);
        free(C8);
        free(B_T);
        free(B_T_T);
        free(A_T);

      }
    }
  }


  printf("All Checks successfull\n");



  return EXIT_SUCCESS;
}
